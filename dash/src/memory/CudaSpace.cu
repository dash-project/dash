#include <dash/Exception.h>
#include <dash/internal/Logging.h>
#include <dash/memory/MemorySpaceBase.h>
#include <dash/memory/MemorySpace.h>
#include <dash/memory/CudaSpace.h>
#include <new>
#include <assert.h>

void* dash::CudaSpace::do_allocate(size_t bytes, size_t alignment)
{
  // Cuda guarantees alignment at 256 bytes but not more.
  assert(alignment <= 256);
  void_pointer ptr;
  auto ret = hipMallocManaged(&ptr, bytes) ;
  if (ret != hipSuccess) {
    DASH_LOG_ERROR(
        "CudaPace.do_allocate",
        "Cannot allocate managed memory",
        bytes,
        alignment);
    DASH_LOG_ERROR("CudaPace.do_allocate", hipGetErrorString(ret));

    std::bad_alloc();
  }
  return ptr;
}

void dash::CudaSpace::do_deallocate(void* p, size_t bytes, size_t alignment)
{
  if (hipFree(p) != hipSuccess) {
    DASH_LOG_ERROR(
        "CudaPace.do_deallocate",
        "Cannot deallocate managed memory",
        p,
        bytes,
        alignment);
  }
}

bool dash::CudaSpace::do_is_equal(
    std::pmr::memory_resource const& other) const noexcept
{
  const CudaSpace* other_p = dynamic_cast<const CudaSpace*>(&other);

  return nullptr != other_p;
}

template <>
dash::MemorySpace<dash::memory_domain_local, dash::memory_space_cuda_tag>*
dash::get_default_memory_space<dash::memory_domain_local, dash::memory_space_cuda_tag>()
{
  static dash::CudaSpace cuda_space_singleton;
  return &cuda_space_singleton;
}

